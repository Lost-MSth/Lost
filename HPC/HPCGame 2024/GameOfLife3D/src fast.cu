#include <chrono>
#include <cstddef>
#include <cstdint>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <utility>

#include <hip/hip_runtime.h>

__constant__ int N2;
__constant__ int Nm;
__constant__ int N1;
__constant__ int R;

__global__ void evolve_kernel(uint8_t *cell_arr, uint8_t *out_arr) {

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    int x = idx & Nm;
    int y = (idx >> R) & Nm;
    int z = (idx >> R) >> R;

    int lz = ((z + N1 - 1) & Nm) * N2;
    int mz = z * N2;
    int rz = ((z + 1) & Nm) * N2;

    int ly = ((y + N1 - 1) & Nm) * N1;
    int my = y * N1;
    int ry = ((y + 1) & Nm) * N1;

    int lx = (x + N1 - 1) & Nm;
    int mx = x;
    int rx = (x + 1) & Nm;

    int alive_count = cell_arr[lx + ly + lz] + cell_arr[lx + ly + mz] +
                      cell_arr[lx + ly + rz] + cell_arr[lx + my + lz] +
                      cell_arr[lx + my + mz] + cell_arr[lx + my + rz] +
                      cell_arr[lx + ry + lz] + cell_arr[lx + ry + mz] +
                      cell_arr[lx + ry + rz] + cell_arr[mx + ly + lz] +
                      cell_arr[mx + ly + mz] + cell_arr[mx + ly + rz] +
                      cell_arr[mx + my + lz] + cell_arr[mx + my + rz] +
                      cell_arr[mx + ry + lz] + cell_arr[mx + ry + mz] +
                      cell_arr[mx + ry + rz] + cell_arr[rx + ly + lz] +
                      cell_arr[rx + ly + mz] + cell_arr[rx + ly + rz] +
                      cell_arr[rx + my + lz] + cell_arr[rx + my + mz] +
                      cell_arr[rx + my + rz] + cell_arr[rx + ry + lz] +
                      cell_arr[rx + ry + mz] + cell_arr[rx + ry + rz];

    uint8_t alive = cell_arr[idx];

    out_arr[idx] = !((alive == 1 && (alive_count < 5 || alive_count > 7)) ||
                     (alive == 0 && alive_count ^ 6));
}

void evolve(uint8_t *cell_arr, uint8_t *out_arr, int n, int t) {
    uint8_t *_in, *_out;
    

    int N2_ptr = n * n;
    int Nm_ptr = n - 1;
    int n_ptr = n;

    size_t num_elem = n * N2_ptr;

    int r;
    if (n == 256) {
        r = 8;
    } else if (n == 512) {
        r = 9;
    } else if (n == 1024) {
        r = 10;
    } else {
        r = 11;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(N2), &N2_ptr, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(Nm), &Nm_ptr, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(N1), &n_ptr, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(R), &r, sizeof(int));

    hipMallocManaged(&_in, num_elem * sizeof(uint8_t));
    hipMallocManaged(&_out, num_elem * sizeof(uint8_t));

    hipMemcpy(_in, cell_arr, num_elem * sizeof(uint8_t), hipMemcpyHostToDevice);

    int threadsPerBlock = 512;
    int blocks = (num_elem + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (int i = 0; i < t; i++) {
        evolve_kernel<<<blocks, threadsPerBlock, 0, stream>>>(_in, _out);
        std::swap(_in, _out);
    }

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    hipMemcpy(out_arr, _in, num_elem * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(_in);
    hipFree(_out);
}


// void evolve(uint8_t *cell_arr, uint8_t *out_arr, int n, int t) {
//     uint8_t *_in, *_out;
//     size_t num_elem;

//     num_elem = n * n * n;

//     int r;
//     if (n == 256) {
//         r = 8;
//     } else if (n == 512) {
//         r = 9;
//     } else if (n == 1024) {
//         r = 10;
//     } else {
//         r = 11;
//     }

//     cudaMallocManaged(&_in, num_elem * sizeof(uint8_t));
//     cudaMallocManaged(&_out, num_elem * sizeof(uint8_t));

//     for (size_t i = 0; i < num_elem; i++) {
//         _in[i] = cell_arr[i];
//     }

//     int threadsPerBlock = 512;
//     int blocks = (num_elem + 1) / threadsPerBlock;

//     for (int i = 0; i < t; i++) {
//         evolve_kernel<<<blocks, threadsPerBlock>>>(_in, _out, n, r);
//         std::swap(_in, _out);
//     }

//     std::swap(_in, _out);

//     cudaDeviceSynchronize();

//     cudaFree(_in);

//     for (size_t i = 0; i < num_elem; i++) {
//         out_arr[i] = _out[i];
//     }

//     cudaFree(_out);
// }

namespace fs = std::filesystem;

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::cout << "Usage: " << argv[0] << " <input_path> <output_path> <N>"
                  << std::endl;
        return 1;
    }

    auto start = std::chrono::high_resolution_clock::now();

    fs::path input_path = argv[1];
    fs::path output_path = argv[2];
    int N = std::atoi(argv[3]);

    size_t M, T;
    std::ifstream input_file(input_path, std::ios::binary);
    input_file.read(reinterpret_cast<char *>(&M), sizeof(M));
    input_file.read(reinterpret_cast<char *>(&T), sizeof(T));

    uint8_t *curr_space = new uint8_t[M * M * M],
            *next_space = new uint8_t[M * M * M];

    input_file.read(reinterpret_cast<char *>(curr_space), M * M * M);

    evolve(curr_space, next_space, M, N);

    T += N;
    std::ofstream output_file(output_path, std::ios::binary);
    output_file.write(reinterpret_cast<char *>(&M), sizeof(M));
    output_file.write(reinterpret_cast<char *>(&T), sizeof(T));
    output_file.write(reinterpret_cast<char *>(next_space), M * M * M);

    delete[] curr_space;
    delete[] next_space;

    auto end = std::chrono::high_resolution_clock::now();
    printf(
        "Time taken: %f\n",
        std::chrono::duration_cast<std::chrono::duration<double>>(end - start)
            .count());

    return 0;
}
