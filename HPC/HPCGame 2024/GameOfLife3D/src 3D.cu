#include <hip/hip_runtime.h>

#include <chrono>
#include <cstddef>
#include <cstdint>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <utility>

__constant__ int N2;
__constant__ int Nm;
__constant__ int N1;
__constant__ int R;

__device__ inline int get_idx(int x, int y, int z) {
    int xx = (x + N1) & Nm;
    int yy = (y + N1) & Nm;
    int zz = (z + N1) & Nm;
    return zz * N2 + yy * N1 + xx;
}

__global__ void evolve_kernel(uint8_t *cell_arr, uint8_t *out_arr) {
    // size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // N2 = N * N

    int thx = threadIdx.x;
    int thy = threadIdx.y;
    int thz = threadIdx.z;

    int x = blockIdx.x * 8 + thx - 1;
    int y = blockIdx.y * 8 + thy - 1;
    int z = blockIdx.z * 8 + thz - 1;
    // size_t idx = x * N2 + y * N1 + z;
    size_t idx = get_idx(x, y, z);

    __shared__ uint8_t cell_arr_shared[1000];
    uint8_t alive = cell_arr[idx];
    int mx = thx + 1;
    int my = (thy + 1) * 10;
    int mz = (thz + 1) * 100;

    cell_arr_shared[mx + my + mz] = alive;

    __syncthreads();

    int lx = thx;
    int rx = thx + 2;
    int ly = my - 10;
    int ry = my + 10;
    int lz = mz - 100;
    int rz = mz + 100;

    if (thx >= 1 && thx <= 8 && thy >= 1 && thy <= 8 && thz >= 1 && thz <= 8) {
        int alive_count =
            cell_arr_shared[lx + ly + lz] + cell_arr_shared[lx + ly + mz] +
            cell_arr_shared[lx + ly + rz] + cell_arr_shared[lx + my + lz] +
            cell_arr_shared[lx + my + mz] + cell_arr_shared[lx + my + rz] +
            cell_arr_shared[lx + ry + lz] + cell_arr_shared[lx + ry + mz] +
            cell_arr_shared[lx + ry + rz] + cell_arr_shared[mx + ly + lz] +
            cell_arr_shared[mx + ly + mz] + cell_arr_shared[mx + ly + rz] +
            cell_arr_shared[mx + my + lz] + cell_arr_shared[mx + my + rz] +
            cell_arr_shared[mx + ry + lz] + cell_arr_shared[mx + ry + mz] +
            cell_arr_shared[mx + ry + rz] + cell_arr_shared[rx + ly + lz] +
            cell_arr_shared[rx + ly + mz] + cell_arr_shared[rx + ly + rz] +
            cell_arr_shared[rx + my + lz] + cell_arr_shared[rx + my + mz] +
            cell_arr_shared[rx + my + rz] + cell_arr_shared[rx + ry + lz] +
            cell_arr_shared[rx + ry + mz] + cell_arr_shared[rx + ry + rz];

        out_arr[idx] = !((alive == 1 && (alive_count < 5 || alive_count > 7)) ||
                         (alive == 0 && alive_count ^ 6));
    }
}

void evolve(uint8_t *cell_arr, uint8_t *out_arr, int n, int t) {
    uint8_t *_in, *_out;

    int N2_ptr = n * n;
    int Nm_ptr = n - 1;
    int n_ptr = n;

    size_t num_elem = n * N2_ptr;

    int r;
    if (n == 256) {
        r = 8;
    } else if (n == 512) {
        r = 9;
    } else if (n == 1024) {
        r = 10;
    } else {
        r = 11;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(N2), &N2_ptr, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(Nm), &Nm_ptr, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(N1), &n_ptr, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(R), &r, sizeof(int));

    hipMallocManaged(&_in, num_elem * sizeof(uint8_t));
    hipMallocManaged(&_out, num_elem * sizeof(uint8_t));

    hipMemcpy(_in, cell_arr, num_elem * sizeof(uint8_t),
               hipMemcpyHostToDevice);

    dim3 threadsPerBlock(10, 10, 10);
    dim3 blocks(n / 8, n / 8, n / 8);

    // int threadsPerBlock = 512;
    // int blocks = (num_elem + threadsPerBlock - 1) / threadsPerBlock;

    size_t shared_mem = sizeof(uint8_t) * 1000;

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (int i = 0; i < t; i++) {
        evolve_kernel<<<blocks, threadsPerBlock, shared_mem, stream>>>(_in,
                                                                       _out);
        std::swap(_in, _out);
    }

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    hipMemcpy(out_arr, _in, num_elem * sizeof(uint8_t),
               hipMemcpyDeviceToHost);

    hipFree(_in);
    hipFree(_out);
}

namespace fs = std::filesystem;

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::cout << "Usage: " << argv[0] << " <input_path> <output_path> <N>"
                  << std::endl;
        return 1;
    }

    auto start = std::chrono::high_resolution_clock::now();

    fs::path input_path = argv[1];
    fs::path output_path = argv[2];
    int N = std::atoi(argv[3]);

    size_t M, T;
    std::ifstream input_file(input_path, std::ios::binary);
    input_file.read(reinterpret_cast<char *>(&M), sizeof(M));
    input_file.read(reinterpret_cast<char *>(&T), sizeof(T));

    uint8_t *curr_space = new uint8_t[M * M * M],
            *next_space = new uint8_t[M * M * M];

    input_file.read(reinterpret_cast<char *>(curr_space), M * M * M);

    evolve(curr_space, next_space, M, N);

    T += N;
    std::ofstream output_file(output_path, std::ios::binary);
    output_file.write(reinterpret_cast<char *>(&M), sizeof(M));
    output_file.write(reinterpret_cast<char *>(&T), sizeof(T));
    output_file.write(reinterpret_cast<char *>(next_space), M * M * M);

    delete[] curr_space;
    delete[] next_space;

    auto end = std::chrono::high_resolution_clock::now();
    printf(
        "Time taken: %f\n",
        std::chrono::duration_cast<std::chrono::duration<double>>(end - start)
            .count());

    return 0;
}
