// #include <math.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdint.h>
#include <stdio.h>

typedef double d_t;
struct d3_t {
    d_t x, y, z;
};

__device__ __host__ inline d_t norm(d3_t x) {
    // return norm3d(x.x, x.y, x.z);
    return sqrt(x.x * x.x + x.y * x.y + x.z * x.z);
}

__device__ __host__ inline d3_t operator-(d3_t a, d3_t b) {
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__global__ void compute(d3_t* __restrict__ mir, d3_t* __restrict__ sen,
                        d_t* __restrict__ d_src_mirn_norm,
                        d_t* __restrict__ data, int64_t mirn, int64_t senn) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if (i >= senn) return;

    d3_t sen_i = sen[i];

    d_t a = 0;
    d_t b = 0;
    d_t tmp_sin = 0;
    d_t tmp_cos = 0;
    for (int64_t j = 0; j < mirn; j++) {
        // d_t l = norm(mir[j] - src) + norm(mir[j] - sen_i);
        d_t l = d_src_mirn_norm[j] + norm(mir[j] - sen_i);
        // d_t tmp = 4000 * l;
        // a += cospi(tmp);
        // b += sinpi(tmp);
        sincospi(4000 * l, &tmp_sin, &tmp_cos);
        a += tmp_cos;
        b += tmp_sin;
    }
    // data[i] = sqrt(a * a + b * b);
    data[i] = hypot(a, b);
}

__global__ void symmetrize(d_t* __restrict__ data, int64_t half_senn) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + half_senn;
    // if (i >= half_senn * 2) return;

    // 假设 sen 有对称性，只计算一半

    // int senn_row = sqrt(half_senn * 2);
    const int64_t senn_row = 1024;
    int64_t j = i % senn_row;
    int64_t k = i / senn_row + 1;

    data[i] = data[half_senn * 2 + j - k * senn_row];
    return;
}

int main() {
    FILE* fi;
    fi = fopen("in.data", "rb");
    d3_t src;
    int64_t mirn, senn;
    d3_t *mir, *sen;

    fread(&src, 1, sizeof(d3_t), fi);

    fread(&mirn, 1, sizeof(int64_t), fi);
    mir = (d3_t*)malloc(mirn * sizeof(d3_t));
    fread(mir, 1, mirn * sizeof(d3_t), fi);

    fread(&senn, 1, sizeof(int64_t), fi);
    sen = (d3_t*)malloc(senn * sizeof(d3_t));
    fread(sen, 1, senn * sizeof(d3_t), fi);

    fclose(fi);

    d_t* data = (d_t*)malloc(senn * sizeof(d_t));

    d3_t *d_mir, *d_sen;
    d_t* d_data;
    d_t* src_mirn_norm = (d_t*)malloc(mirn * sizeof(d_t));

    for (int64_t i = 0; i < mirn; i++) {
        src_mirn_norm[i] = norm(mir[i] - src);
    }

    d_t* d_src_mirn_norm;

    hipMalloc(&d_mir, mirn * sizeof(d3_t));
    hipMalloc(&d_sen, senn * sizeof(d3_t));
    hipMalloc(&d_data, senn * sizeof(d_t));
    hipMalloc(&d_src_mirn_norm, mirn * sizeof(d_t));

    hipMemcpy(d_mir, mir, mirn * sizeof(d3_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sen, sen, senn * sizeof(d3_t), hipMemcpyHostToDevice);
    hipMemcpy(d_src_mirn_norm, src_mirn_norm, mirn * sizeof(d_t),
               hipMemcpyHostToDevice);

    int blockSize = 1024;
    int numBlocks = (senn / 2 + blockSize - 1) / blockSize;

    fprintf(stderr, "numBlocks = %d\n", numBlocks);
    fprintf(stderr, "blockSize = %d\n", blockSize);

    compute<<<numBlocks, blockSize>>>(d_mir, d_sen, d_src_mirn_norm, d_data,
                                      mirn, senn / 2);
    symmetrize<<<numBlocks, blockSize>>>(d_data, senn / 2);

    hipMemcpy(data, d_data, senn * sizeof(d_t), hipMemcpyDeviceToHost);

    hipFree(d_mir);
    hipFree(d_sen);
    hipFree(d_data);
    hipFree(d_src_mirn_norm);

    fi = fopen("out.data", "wb");
    fwrite(data, 1, senn * sizeof(d_t), fi);
    fclose(fi);

    free(mir);
    free(sen);
    free(data);
    free(src_mirn_norm);

    return 0;
}