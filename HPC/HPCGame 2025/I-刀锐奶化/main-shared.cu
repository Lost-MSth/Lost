#include "hip/hip_runtime.h"
// #include <math.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdint.h>
#include <stdio.h>

typedef double d_t;
struct d3_t {
    d_t x, y, z;
};

__device__ __host__ inline d_t norm(d3_t x) { 
    // return norm3d(x.x, x.y, x.z);
    return sqrt(x.x * x.x + x.y * x.y + x.z * x.z); 
    }

__device__ __host__ inline d3_t operator-(d3_t a, d3_t b) {
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}


__global__ void compute(d3_t* __restrict__ mir, d3_t* __restrict__ sen, d_t * __restrict__ d_src_mirn_norm, d_t* __restrict__ data, int64_t mirn,
                        int64_t senn) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= senn) return;

    extern __shared__ d3_t shared[1024];

    d3_t sen_i = sen[i];

    d_t a = 0;
    d_t b = 0;
    d_t tmp_sin = 0;
    d_t tmp_cos = 0;

    int times = mirn / blockDim.x;

    for (int t = 0; t < times; ++t) {
        shared[threadIdx.x] = mir[t * blockDim.x + threadIdx.x];
        __syncthreads();

        #pragma unroll
        for (int j = 0; j < blockDim.x; ++j) {
            d_t l = d_src_mirn_norm[t * blockDim.x + j] + norm(shared[j] - sen_i);
            sincospi(4000 * l, &tmp_sin, &tmp_cos);
            a += tmp_cos;
            b += tmp_sin;
        }
        __syncthreads();
    }
    // for (int64_t j = 0; j < mirn; j++) {
    //     // d_t l = norm(mir[j] - src) + norm(mir[j] - sen_i);
    //     d_t l = d_src_mirn_norm[j] + norm(mir[j] - sen_i);
    //     // d_t tmp = 4000 * l;
    //     // a += cospi(tmp);
    //     // b += sinpi(tmp);
        
    //     sincospi(4000 * l, &tmp_sin, &tmp_cos);

    //     // printf("l = %f tmp_sin = %f tmp_cos = %f\n", l, tmp_sin, tmp_cos);

    //     a += tmp_cos;
    //     b += tmp_sin;
    // }
    // data[i] = sqrt(a * a + b * b);
    data[i] = hypot(a, b);
}

#define CONST 602

int main() {
    FILE* fi;
    fi = fopen("in.data", "rb");
    d3_t src;
    int64_t mirn, senn;
    d3_t *mir, *sen;

    fread(&src, 1, sizeof(d3_t), fi);

    fread(&mirn, 1, sizeof(int64_t), fi);
    mir = (d3_t*)malloc(mirn * sizeof(d3_t));
    fread(mir, 1, mirn * sizeof(d3_t), fi);

    fread(&senn, 1, sizeof(int64_t), fi);
    sen = (d3_t*)malloc(senn * sizeof(d3_t));
    fread(sen, 1, senn * sizeof(d3_t), fi);

    fclose(fi);

    d_t* data = (d_t*)malloc(senn * sizeof(d_t));

    d3_t *d_mir, *d_sen;
    d_t* d_data;
    d_t* src_mirn_norm = (d_t*)malloc(mirn * sizeof(d_t));

    for (int64_t i = 0; i < mirn; i++) {
        src_mirn_norm[i] = norm(mir[i] - src) - CONST;
    }

    d_t* d_src_mirn_norm;

    hipMalloc(&d_mir, mirn * sizeof(d3_t));
    hipMalloc(&d_sen, senn * sizeof(d3_t));
    hipMalloc(&d_data, senn * sizeof(d_t));
    hipMalloc(&d_src_mirn_norm, mirn * sizeof(d_t));

    hipMemcpy(d_mir, mir, mirn * sizeof(d3_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sen, sen, senn * sizeof(d3_t), hipMemcpyHostToDevice);
    hipMemcpy(d_src_mirn_norm, src_mirn_norm, mirn * sizeof(d_t), hipMemcpyHostToDevice);
 

    int blockSize = 1024;
    int numBlocks = (senn + blockSize - 1) / blockSize;

    fprintf(stderr, "numBlocks = %d\n", numBlocks);
    fprintf(stderr, "blockSize = %d\n", blockSize);

    compute<<<numBlocks, blockSize, 1024 * sizeof(d3_t)>>>(d_mir, d_sen, d_src_mirn_norm, d_data, mirn, senn);

    hipMemcpy(data, d_data, senn * sizeof(d_t), hipMemcpyDeviceToHost);

    hipFree(d_mir);
    hipFree(d_sen);
    hipFree(d_data);
    hipFree(d_src_mirn_norm);

    fi = fopen("out.data", "wb");
    fwrite(data, 1, senn * sizeof(d_t), fi);
    fclose(fi);

    free(mir);
    free(sen);
    free(data);
    free(src_mirn_norm);

    return 0;
}